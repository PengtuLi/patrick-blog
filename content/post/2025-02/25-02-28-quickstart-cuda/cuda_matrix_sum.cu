#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<sys/time.h>
#define CHECK(call)                     \
{                                       \
    const hipError_t error = call;     \
    if(error!=hipSuccess)              \
    {                                   \
        printf("Error: %s:%d",__FILE__,__LINE__);      \
        std::cout<<"code: "<<error<<" ,reason: "<<hipGetErrorString(error)<<std::endl;     \
        exit(-10*error);     \
    }                        \
}
//使用gettimeofday会获取自1970年1月1日0点以来到现在的秒数
//timeval是一个结构体，其中有成员 tv_sec:秒 tv_usec:微秒
double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}
//初始化数组
void initialData(float *ip,int size)
{
    //generate different seed for random number
    time_t t;
    srand((unsigned)time(&t));
    for(int i=0;i<size;i++)
    {
        ip[i]=i;
    }
}
//hostRef传入CPU端计算的矩阵加法结果，gpuRef传入GPU端计算的矩阵加法结果
//对比争取输出"Arrats match"
void checkResult(float *hostRef,float *gpuRef,const int N)
{
    double epsilon = 1.0E-8;
    bool match=1;
    for(int i=0;i<N;i++)
    {
        if(abs(hostRef[i]-gpuRef[i])>epsilon)
        {
            match=0;
            printf("Arrays do not match");
            printf("host %5.2f gpu %5.2f at current %d\n",hostRef[i],gpuRef[i],i);
            break;
        }
    }
    if(match)
      std::cout<<"Arrats match"<<std::endl;
}
//cpu端计算矩阵加法
void sumMatrixOnHost(float *A,float *B,float *C,const int nx ,const int ny)
{
    float *ia=A;
    float *ib=B;
    float *ic=C;
    for(int iy=0;iy<ny;iy++)
    {
        for(int ix=0;ix<nx;ix++)
        {
            ic[ix]=ia[ix]+ib[ix];
        }
        ia += nx;
        ib += nx;
        ic += nx;
    }
}
//cuda核函数计算矩阵加法
__global__ void sumMatrixOnGPU(float *MatA,float *MatB,float *MatC,int nx,int ny)
{
    //使用前问中的线程全局索引的计算方式
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy*nx + ix;
    if(ix<nx && iy<ny)
    {
        //这种线程的全局索引方式正好是与按行优先的存储的矩阵的索引方式是一致的
        //所以线程的全局索引可以与矩阵中元素的索引很好的对应
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}
int main(int argc,char **argv)
{
    int dev = 0;
    hipDeviceProp_t deviceProp;
    //CHECK宏定义检查操作是否正常处理
    CHECK(hipGetDeviceProperties(&deviceProp,dev));
    printf("Using Device %d: %s\n",dev,deviceProp.name);
    CHECK(hipSetDevice(dev));
    //set up data size of matrix
    int nx = 1<<14; //16384
    int ny = 1<<14; //16384
    int nxy = nx*ny;
    int nBytes = nxy*sizeof(float);
    printf("Matrix size: nx %d ny %d\n",nx,ny);
    //malloc host memory
    float *h_A,*h_B,*hostRef,*gpuRef;
    h_A = (float*)malloc(nBytes);
    h_B = (float*)malloc(nBytes);
    hostRef = (float*)malloc(nBytes);
    gpuRef = (float*)malloc(nBytes);
    //init data at host side
    double iStart = cpuSecond();
    initialData(h_A,nxy);
    initialData(h_B,nxy);
    double iElaps = cpuSecond() - iStart;
    memset(hostRef,0,nBytes);
    memset(gpuRef,0,nBytes);
    iStart = cpuSecond();
    sumMatrixOnHost(h_A,h_B,hostRef,nx,ny);
    iElaps = cpuSecond() - iStart; //cpu 端耗时
    std::cout<<"sumMatrixOnHost cost "<<iElaps<<"sec\n";
    //malloc device global memory
    //GPU 申请GPU端空间
    float *d_MatA,*d_MatB,*d_MatC;
    hipMalloc((void**)&d_MatA,nBytes);
    hipMalloc((void**)&d_MatB,nBytes);
    hipMalloc((void**)&d_MatC,nBytes);
    //transfer data from host to device
    //数据传输
    hipMemcpy(d_MatA,h_A,nBytes,hipMemcpyHostToDevice);
    hipMemcpy(d_MatB,h_B,nBytes,hipMemcpyHostToDevice);
    //invoke kernel at host side
    int dimx = 32;
    int dimy = 32;
    //block size = (32,32)
    //也就是每个block中有32*32个线程（结构是二维）
    dim3 block(dimx,dimy);
    //grid size = (512,512)
    //也就是每个grid中有512*512个block （结构是二维）
    dim3 grid((nx+block.x-1)/block.x,((ny+block.y-1)/block.y));
    iStart = cpuSecond();//gpu初始时间
    sumMatrixOnGPU<<<grid,block>>>(d_MatA,d_MatB,d_MatC,nx,ny);//以上述配置线程层级结构的方式启动核函数
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    printf("sumMatrixOnGPU<<<(%d,%d),(%d,%d)>>>elapsed %f sec\n",grid.x,grid.y,block.x,block.y,iElaps);
    //copy kernel result back to host side
    //再把GPU计算的结果拷贝会cpu端
    hipMemcpy(gpuRef,d_MatC,nBytes,hipMemcpyDeviceToHost);
    //check device res
    checkResult(hostRef,gpuRef,nxy);
    //释放gpu中申请的内存
    hipFree(d_MatA);
    hipFree(d_MatB);
    hipFree(d_MatC);
    //释放主机端内存
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);
    //reset device 
    hipDeviceReset();
    return (0);
}
